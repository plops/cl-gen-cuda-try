#include "hip/hip_runtime.h"
#include <stdio.h>
// https://www.youtube.com/watch?v=Ed_h2km0liI CUDACast #2 - Your First CUDA C
// Program
// https://github.com/NVIDIA-developer-blog/cudacasts/blob/master/ep2-first-cuda-c-program/kernel.cu
__global__ void vector_add(int *a, int *b, int *c, int n) {
  {
    int i = threadIdx.x;
    if ((i < n)) {
      c[i] = (a[i] + b[i]);
    }
  }
}
enum { N = 1024 };

int main() {
  {
    int *a = malloc((N * sizeof(int)));
    int *b = malloc((N * sizeof(int)));
    int *c = malloc((N * sizeof(int)));
    int *d_a;
    int *d_b;
    int *d_c;
    hipMalloc(&d_a, (N * sizeof(int)));
    hipMalloc(&d_b, (N * sizeof(int)));
    hipMalloc(&d_c, (N * sizeof(int)));
    for (unsigned int i = 0; (i < N); i += 1) {
      a[i] = i;
      b[i] = i;
      c[i] = 0;
    }
    hipMemcpy(d_a, a, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, (N * sizeof(int)), hipMemcpyHostToDevice);
    vector_add<<<1, N>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, (N * sizeof(int)), hipMemcpyDeviceToHost);
    free(a);
    hipFree(d_a);
    free(b);
    hipFree(d_b);
    free(c);
    hipFree(d_c);
    return 0;
  }
}