#include "hip/hip_runtime.h"
#include <stdio.h>
// https://www.youtube.com/watch?v=Ed_h2km0liI CUDACast #2 - Your First CUDA C
// Program
// https://github.com/NVIDIA-developer-blog/cudacasts/blob/master/ep2-first-cuda-c-program/kernel.cu
__global__ void vector_add(int *a, int *b, int *c, int n) {
  {
    int i = threadIdx.x;
    if ((i < n)) {
      c[i] = (a[i] + b[i]);
    }
  }
}
enum { N = 1024 };

int main() {
  {
    int cuda_dev;
    hipChooseDevice(&cuda_dev, NULL);
    // read device attributes;
    {
      int val;
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxThreadsPerBlock, cuda_dev);
      printf("hipDeviceAttributeMaxThreadsPerBlock=%d (Maximum number of threads per "
             "block)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxBlockDimX, cuda_dev);
      printf("hipDeviceAttributeMaxBlockDimX=%d (Maximum x-dimension of a block)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxBlockDimY, cuda_dev);
      printf("hipDeviceAttributeMaxBlockDimY=%d (Maximum y-dimension of a block)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxBlockDimZ, cuda_dev);
      printf("hipDeviceAttributeMaxBlockDimZ=%d (Maximum z-dimension of a block)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxGridDimX, cuda_dev);
      printf("hipDeviceAttributeMaxGridDimX=%d (Maximum x-dimension of a grid)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxGridDimY, cuda_dev);
      printf("hipDeviceAttributeMaxGridDimY=%d (Maximum y-dimension of a grid)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxGridDimZ, cuda_dev);
      printf("hipDeviceAttributeMaxGridDimZ=%d (Maximum z-dimension of a grid)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSharedMemoryPerBlock,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSharedMemoryPerBlock=%d (Maximum amount of shared "
             "memoryavailable to a thread block in bytes)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeTotalConstantMemory, cuda_dev);
      printf("hipDeviceAttributeTotalConstantMemory=%d (Memory available on device "
             "for __constant__variables in a CUDA C kernel in bytes)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeWarpSize, cuda_dev);
      printf("hipDeviceAttributeWarpSize=%d (Warp size in threads)\n", val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxPitch, cuda_dev);
      printf("hipDeviceAttributeMaxPitch=%d (Maximum pitch in bytes allowed by the "
             "memory copyfunctions that involve memory regions allocated "
             "through hipMallocPitch())\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture1DWidth, cuda_dev);
      printf("hipDeviceAttributeMaxTexture1DWidth=%d (Maximum 1D texture width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture1DLinear,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture1DLinear=%d (Maximum width for a 1D "
             "texture boundto linear memory)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture1DMipmap,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture1DMipmap=%d (Maximum mipmapped 1D "
             "texturewidth)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DWidth, cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DWidth=%d (Maximum 2D texture width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DHeight, cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DHeight=%d (Maximum 2D texture height)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DLinear,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DLinear=%d (Maximum width for a 2D "
             "texture boundto linear memory)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DLinear,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DLinear=%d (Maximum height for a 2D "
             "texture boundto linear memory)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DLinear,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DLinear=%d (Maximum pitch in bytes "
             "for a 2D texturebound to linear memory)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DMipmap,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DMipmap=%d (Maximum mipmapped 2D "
             "texturewidth)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DMipmap,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DMipmap=%d (Maximum mipmapped 2D "
             "textureheight)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DWidth, cuda_dev);
      printf("hipDeviceAttributeMaxTexture3DWidth=%d (Maximum 3D texture width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DHeight, cuda_dev);
      printf("hipDeviceAttributeMaxTexture3DHeight=%d (Maximum 3D texture height)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DDepth, cuda_dev);
      printf("hipDeviceAttributeMaxTexture3DDepth=%d (Maximum 3D texture depth)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DAlt, cuda_dev);
      printf("hipDeviceAttributeMaxTexture3DAlt=%d (Alternate maximum 3D texture "
             "width, 0 if noalternate maximum 3D texture size is supported)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DAlt, cuda_dev);
      printf(
          "hipDeviceAttributeMaxTexture3DAlt=%d (Alternate maximum 3D texture "
          "height, 0 ifno alternate maximum 3D texture size is supported)\n",
          val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture3DAlt, cuda_dev);
      printf("hipDeviceAttributeMaxTexture3DAlt=%d (Alternate maximum 3D texture "
             "depth, 0 if noalternate maximum 3D texture size is supported)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTextureCubemap, cuda_dev);
      printf("hipDeviceAttributeMaxTextureCubemap=%d (Maximum cubemap texture "
             "width orheight)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture1DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture1DLayered=%d (Maximum 1D layered "
             "texture width)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxTexture1DLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxTexture1DLayeredLayers=%d (Maximum layers in a 1D "
             "layeredtexture)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DLayered=%d (Maximum 2D layered "
             "texture width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTexture2DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTexture2DLayered=%d (Maximum 2D layered "
             "texture height)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxTexture2DLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxTexture2DLayeredLayers=%d (Maximum layers in a 2D "
             "layeredtexture)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxTextureCubemapLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxTextureCubemapLayered=%d (Maximum cubemap "
             "layeredtexture width or height)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxTextureCubemapLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxTextureCubemapLayeredLayers=%d (Maximum layers in "
             "a cubemaplayered texture)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface1D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface1D=%d (Maximum 1D surface width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface2D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface2D=%d (Maximum 2D surface width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface2D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface2D=%d (Maximum 2D surface height)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface3D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface3D=%d (Maximum 3D surface width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface3D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface3D=%d (Maximum 3D surface height)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface3D, cuda_dev);
      printf("hipDeviceAttributeMaxSurface3D=%d (Maximum 3D surface depth)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface1DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSurface1DLayered=%d (Maximum 1D layered "
             "surface width)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxSurface1DLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxSurface1DLayeredLayers=%d (Maximum layers in a 1D "
             "layeredsurface)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface2DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSurface2DLayered=%d (Maximum 2D layered "
             "surface width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurface2DLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSurface2DLayered=%d (Maximum 2D layered "
             "surface height)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxSurface2DLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxSurface2DLayeredLayers=%d (Maximum layers in a 2D "
             "layeredsurface)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurfaceCubemap, cuda_dev);
      printf("hipDeviceAttributeMaxSurfaceCubemap=%d (Maximum cubemap surface "
             "width)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSurfaceCubemapLayered,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSurfaceCubemapLayered=%d (Maximum cubemap "
             "layeredsurface width)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrMaxSurfaceCubemapLayeredLayers,
                             cuda_dev);
      printf("cudaDevAttrMaxSurfaceCubemapLayeredLayers=%d (Maximum layers in "
             "a cubemaplayered surface)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxRegistersPerBlock, cuda_dev);
      printf("hipDeviceAttributeMaxRegistersPerBlock=%d (Maximum number of 32-bit "
             "registers availableto a thread block)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeClockRate, cuda_dev);
      printf("hipDeviceAttributeClockRate=%d (Peak clock frequency in kilohertz)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeTextureAlignment, cuda_dev);
      printf("hipDeviceAttributeTextureAlignment=%d (Alignment requirement texture "
             "base addressesaligned to textureAlign bytes do not need an "
             "offset applied to texture fetches)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeTexturePitchAlignment, cuda_dev);
      printf("hipDeviceAttributeTexturePitchAlignment=%d (Pitch alignment requirement "
             "for 2D texturereferences bound to pitched memory)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeAsyncEngineCount, cuda_dev);
      printf("hipDeviceAttributeAsyncEngineCount=%d (1 if the device can concurrently copy "
             "memory betweenhost and device while executing a kernel, or 0 if "
             "not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, cuda_dev);
      printf("hipDeviceAttributeMultiprocessorCount=%d (Number of multiprocessors on "
             "the device)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeKernelExecTimeout, cuda_dev);
      printf("hipDeviceAttributeKernelExecTimeout=%d (1 if there is a run time limit "
             "for kernels executedon the device, or 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeIntegrated, cuda_dev);
      printf("hipDeviceAttributeIntegrated=%d (1 if the device is integrated with the "
             "memory subsystem, or0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeCanMapHostMemory, cuda_dev);
      printf("hipDeviceAttributeCanMapHostMemory=%d (1 if the device can map host "
             "memory into theCUDA address space, or 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeComputeMode, cuda_dev);
      printf("hipDeviceAttributeComputeMode=%d (Compute mode is the compute mode that "
             "the device iscurrently in. Available modes are as follows)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeConcurrentKernels, cuda_dev);
      printf("hipDeviceAttributeConcurrentKernels=%d (1 if the device supports "
             "executing multiple kernelswithin the same context "
             "simultaneously, or 0 if not. It is not guaranteed that "
             "multipkernels will be resident on the device concurrently so "
             "this feature should not berelied upon for correctness)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeEccEnabled, cuda_dev);
      printf("hipDeviceAttributeEccEnabled=%d (1 if error correction is enabled on "
             "the device, 0 if errorcorrection is disabled or not supported by "
             "the device)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributePciBusId, cuda_dev);
      printf("hipDeviceAttributePciBusId=%d (PCI bus identifier of the device)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributePciDeviceId, cuda_dev);
      printf("hipDeviceAttributePciDeviceId=%d (PCI device (also known as slot) "
             "identifier of the device)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeTccDriver, cuda_dev);
      printf("hipDeviceAttributeTccDriver=%d (1 if the device is using a TCC driver. "
             "TCC is only availableon Tesla hardware running Windows Vista or "
             "later)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMemoryClockRate, cuda_dev);
      printf("hipDeviceAttributeMemoryClockRate=%d (Peak memory clock frequency in "
             "kilohertz)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMemoryBusWidth, cuda_dev);
      printf("hipDeviceAttributeMemoryBusWidth=%d (Global memory bus width in "
             "bits)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeL2CacheSize, cuda_dev);
      printf("hipDeviceAttributeL2CacheSize=%d (Size of L2 cache in bytes. 0 if the "
             "device doesn't have L2cache)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxThreadsPerMultiProcessor,
                             cuda_dev);
      printf("hipDeviceAttributeMaxThreadsPerMultiProcessor=%d (Maximum resident "
             "threads permultiprocessor)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeUnifiedAddressing, cuda_dev);
      printf("hipDeviceAttributeUnifiedAddressing=%d (1 if the device shares a "
             "unified address space withthe host, or 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMajor, cuda_dev);
      printf("hipDeviceAttributeComputeCapabilityMajor=%d (Major compute capability "
             "version number)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMinor, cuda_dev);
      printf("hipDeviceAttributeComputeCapabilityMinor=%d (Minor compute capability "
             "version number)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeStreamPrioritiesSupported,
                             cuda_dev);
      printf("hipDeviceAttributeStreamPrioritiesSupported=%d (1 if the device "
             "supports stream priorities, or0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeGlobalL1CacheSupported, cuda_dev);
      printf("hipDeviceAttributeGlobalL1CacheSupported=%d (1 if device supports "
             "caching globals in L1cache, 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeLocalL1CacheSupported, cuda_dev);
      printf("hipDeviceAttributeLocalL1CacheSupported=%d (1 if device supports "
             "caching locals in L1cache, 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                             cuda_dev);
      printf("hipDeviceAttributeMaxSharedMemoryPerMultiprocessor=%d (Maximum amount "
             "of sharedmemory available to a multiprocessor in bytes this "
             "amount is shared by all threadblocks simultaneously resident on "
             "a multiprocessor)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMaxRegistersPerMultiprocessor,
                             cuda_dev);
      printf(
          "hipDeviceAttributeMaxRegistersPerMultiprocessor=%d (Maximum number of "
          "32-bitregisters available to a multiprocessor this number is shared "
          "by all thread blockssimultaneously resident on a multiprocessor)\n",
          val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeManagedMemory, cuda_dev);
      printf("hipDeviceAttributeManagedMemory=%d (1 if device supports allocating "
             "managed memory, 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeIsMultiGpuBoard, cuda_dev);
      printf("hipDeviceAttributeIsMultiGpuBoard=%d (1 if device is on a multi-GPU "
             "board, 0 if not)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeMultiGpuBoardGroupID, cuda_dev);
      printf("hipDeviceAttributeMultiGpuBoardGroupID=%d (Unique identifier for a "
             "group of devices onthe same multi-GPU board)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeHostNativeAtomicSupported,
                             cuda_dev);
      printf("hipDeviceAttributeHostNativeAtomicSupported=%d (1 if the link between "
             "the device and thehost supports native atomic operations)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeSingleToDoublePrecisionPerfRatio,
                             cuda_dev);
      printf("hipDeviceAttributeSingleToDoublePrecisionPerfRatio=%d (Ratio of single "
             "precisionperformance (in floating-point operations per second) "
             "to double precision performance)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributePageableMemoryAccess, cuda_dev);
      printf("hipDeviceAttributePageableMemoryAccess=%d (1 if the device supports "
             "coherently accessingpageable memory without calling "
             "hipHostRegister on it, and 0 otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeConcurrentManagedAccess,
                             cuda_dev);
      printf("hipDeviceAttributeConcurrentManagedAccess=%d (1 if the device can "
             "coherently accessmanaged memory concurrently with the CPU, and 0 "
             "otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeComputePreemptionSupported,
                             cuda_dev);
      printf("hipDeviceAttributeComputePreemptionSupported=%d (1 if the device "
             "supports ComputePreemption, 0 if not.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeCanUseHostPointerForRegisteredMem,
                             cuda_dev);
      printf("hipDeviceAttributeCanUseHostPointerForRegisteredMem=%d (1 if the device "
             "can access hostregistered memory at the same virtual address as "
             "the CPU, and 0 otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeCooperativeLaunch, cuda_dev);
      printf("hipDeviceAttributeCooperativeLaunch=%d (1 if the device supports "
             "launching cooperativekernels via hipLaunchCooperativeKernel, "
             "and 0 otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeCooperativeMultiDeviceLaunch,
                             cuda_dev);
      printf("hipDeviceAttributeCooperativeMultiDeviceLaunch=%d (1 if the device "
             "supports launchingcooperative kernels via "
             "hipLaunchCooperativeKernelMultiDevice, and 0otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, cudaDevAttrCanFlushRemoteWrites, cuda_dev);
      printf("cudaDevAttrCanFlushRemoteWrites=%d (1 if the device supports "
             "flushing ofoutstanding remote writes, and 0 otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeHostRegisterSupported, cuda_dev);
      printf(
          "hipDeviceAttributeHostRegisterSupported=%d (1 if the device supports host "
          "memoryregistration via hipHostRegister, and 0 otherwise.)\n",
          val);
      hipDeviceGetAttribute(
          &val, hipDeviceAttributePageableMemoryAccessUsesHostPageTables, cuda_dev);
      printf("hipDeviceAttributePageableMemoryAccessUsesHostPageTables=%d (1 if the "
             "device accessespageable memory via the host's page tables, and 0 "
             "otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeDirectManagedMemAccessFromHost,
                             cuda_dev);
      printf("hipDeviceAttributeDirectManagedMemAccessFromHost=%d (1 if the host can "
             "directly accessmanaged memory on the device without migration, "
             "and 0 otherwise.)\n",
             val);
      hipDeviceGetAttribute(&val, hipDeviceAttributeSharedMemPerBlockOptin,
                             cuda_dev);
      printf("hipDeviceAttributeSharedMemPerBlockOptin=%d (Maximum per "
             "blockshared memory size on the device. This value can be opted "
             "into when usingcudaFuncSetAttribute)\n",
             val);
    }
  }
  {
    int *a = static_cast<int *>(malloc((N * sizeof(int))));
    int *b = static_cast<int *>(malloc((N * sizeof(int))));
    int *c = static_cast<int *>(malloc((N * sizeof(int))));
    int *d_a;
    int *d_b;
    int *d_c;
    hipMalloc(&d_a, (N * sizeof(int)));
    hipMalloc(&d_b, (N * sizeof(int)));
    hipMalloc(&d_c, (N * sizeof(int)));
    for (unsigned int i = 0; (i < N); i += 1) {
      a[i] = i;
      b[i] = i;
      c[i] = 0;
    }
    hipMemcpy(d_a, a, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, (N * sizeof(int)), hipMemcpyHostToDevice);
    vector_add<<<1, N>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, (N * sizeof(int)), hipMemcpyDeviceToHost);
    free(a);
    hipFree(d_a);
    free(b);
    hipFree(d_b);
    free(c);
    hipFree(d_c);
    return 0;
  }
}